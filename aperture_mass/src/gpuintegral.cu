#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <math.h>
#include <unistd.h>


#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
//#include "cutil.h"
using namespace std;




int checkDeviceSpecs(int number_of_galaxies, int grid_size);


//KERNEL
__global__ void integrandKernel(double* x, double* funcEval){

  int idx = threadIdx.x;
	   
  funcEval[idx] = sin(x[idx]);


}



double doIntegral(){

  double sum;
  int nevals = 32*10000;
  int i;

  double xlow, xhigh;
  xlow = 0.;
  xhigh = M_PI/2.;

  int sizeneeded = nevals*sizeof(double);
    
  
  //CPU Memory
  double* h_xvals = (double*)malloc(sizeneeded);
  double* h_funcEvals = (double*)malloc(sizeneeded);
  
  
  for (i=0; i < nevals; i++){
    h_xvals[i] = (xhigh - xlow)*i/nevals;
    h_funcEvals[i] = 0.;
  }


  printf("a\n");



  /// first, I need to test whether the device is busy. If so, it can wait a little while.
  while(1){
    size_t testsize = 1*sizeof(float); 
    float *d_test;
    hipMalloc(&d_test, testsize);
    hipError_t err = hipGetLastError();
    if( hipSuccess != err){
      printf("gotta wait for a bit!: %s\n",  hipGetErrorString( err) );
      sleep(10);
    }
    else break;
  }


  printf("a\n");


  //GPU memory
  double *d_xvals, *d_funcEvals;
  hipMalloc(&d_xvals, sizeneeded);
  hipMalloc(&d_funcEvals, sizeneeded);

  hipMemcpy(d_xvals, h_xvals, sizeneeded, hipMemcpyHostToDevice);
  hipMemcpy(d_funcEvals, h_funcEvals, sizeneeded, hipMemcpyHostToDevice);

  int threadsPerBlock = nevals;
  int blocksPerGrid = 1.;

  printf("a\n");

  integrandKernel<<<blocksPerGrid, threadsPerBlock>>>(d_xvals,d_funcEvals );

  printf("a\n");


  hipMemcpy(h_funcEvals, d_funcEvals, sizeneeded, hipMemcpyDeviceToHost);

  printf("a\n");


  for (i=1; i < nevals-1; i++){
    sum += 2*h_funcEvals[i];
    printf("%f\n", h_funcEvals[i]);
  }

  printf("a\n");


  sum += (h_funcEvals[0] + h_funcEvals[nevals-1]);

  sum *= (xhigh - xlow)/(2.*nevals);

  printf("a\n");

  return sum;

}


int main(int argc, char **argv){

  int nloops = atoi(argv[1]);
  int i;
  double sum = 0;

  for (i=0; i < nloops; i++){
    sum = doIntegral();
  
    printf("%f\n", sum);
  }
  

}
